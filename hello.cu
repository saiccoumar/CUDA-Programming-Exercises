
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

__global__ void hello()
{
  printf("Hello\n");
}


int main(void)
{
  int count, device;
  hipGetDeviceCount(&count);
  hipGetDevice(&device);
  printf("You have in total %d devices in your system\n", count);
  printf("GPU %d will now print a message for you:\n", device);

  hello<<<2,2>>>();
  hipDeviceSynchronize();
  
  return 0;	
}